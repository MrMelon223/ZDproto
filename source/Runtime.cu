#include "hip/hip_runtime.h"
	// Runtime.cu
#include "../include/Runtime.h"

std::vector<HostModel> HOST_MODELS;
std::vector<d_Model> DEVICE_MODELS;

std::vector<Object> Runtime::OBJECTS;
std::vector<BulletWeapon> Runtime::WEAPONS;

Object* Runtime::PLAYER_OBJECT;
std::vector<d_ModelInstance> Runtime::model_instances;

int sql_callback(void* p_data, int num_fields, char** p_fields, char** p_col_names)
{
	//try {
		std::cout << "Fields: " << num_fields << std::endl;
		std::cout << "     Loading Asset " << p_fields[0] << std::endl;
		HOST_MODELS.push_back(HostModel(std::string(p_fields[0])));
		DEVICE_MODELS.push_back(HOST_MODELS.back().to_gpu());
	/* }
	catch (...) {
		// abort select on failure, don't let exception propogate thru sqlite3 call-stack
		return 1;
	}*/
	return 0;
}

void Runtime::runtime_load(sqlite3* sql) {
	std::string filepath = "resources/models.txt";

	const char* command = "SELECT path FROM assets;";
	char* err;

	HOST_MODELS = *new std::vector<HostModel>();
	DEVICE_MODELS = *new std::vector<d_Model>();

	int r = sqlite3_exec(sql, command, sql_callback, NULL, &err);
	if (r != SQLITE_OK) {
		std::cerr << "SQL Model error: " << err << std::endl;
		return;
	}

}

int sql_callback_object(void* p_data, int num_fields, char** p_fields, char** p_col_names)
{
	try {
		std::cout << "	Objects:" << std::endl;
			Object o;

			std::string obj_name, vis_model_name, hitbox_model_name;
			uint32_t obj_type;
			float mass_kg;
			int bullet_coll;

			for (size_t i = 0; i < num_fields; i++) {
				char* str = p_fields[i];
				std::istringstream in(str);

				switch (i) {
				case 0:
					in >> obj_name;

					break;
				case 1:
					in >> vis_model_name;
					break;
				case 2:
					in >> hitbox_model_name;
					break;
				case 3:
					in >> obj_type;
					break;
				case 4:
					in >> mass_kg;
					break;
				case 5:
					in >> bullet_coll;
					break;
				}

				std::cout << str << std::endl;
			}
			if (obj_type == 0) {
				ObjIndexs obj_idx;

				d_ModelInstance d_mi = create_instance(Runtime::find_host_model_index(vis_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(vis_model_name)->get_triangle_count(), false, 1.0f);
				Runtime::model_instances.push_back(d_mi);
				uint32_t d_mi_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);
				d_ModelInstance d_hitbox = create_instance(Runtime::find_host_model_index(hitbox_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(hitbox_model_name)->get_triangle_count(), true, 1.0f);
				Runtime::model_instances.push_back(d_hitbox);
				uint32_t d_hitbox_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

				obj_idx.physics_object_index = Runtime::OBJECTS.size() - 1;
				o = Object(ObjectType::AI, obj_idx, obj_name, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f),Runtime::find_host_model_index(vis_model_name),  d_mi_idx, d_hitbox_idx);
				o.set_mass(mass_kg);
				o.set_max_health(100.0f);
				o.set_health(100.0f);
			}
			else if (obj_type == 1) {
				ObjIndexs obj_idx;

				d_ModelInstance d_mi = create_instance(Runtime::find_host_model_index(vis_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(vis_model_name)->get_triangle_count(), false, 1.0f);
				Runtime::model_instances.push_back(d_mi);
				uint32_t d_mi_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);
				d_ModelInstance d_hitbox = create_instance(Runtime::find_host_model_index(hitbox_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(hitbox_model_name)->get_triangle_count(), true, 1.0f);
				Runtime::model_instances.push_back(d_hitbox);
				uint32_t d_hitbox_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

				obj_idx.physics_object_index = Runtime::OBJECTS.size() - 1;
				o = Object(ObjectType::Physics, obj_idx, obj_name, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model_index(vis_model_name), d_mi_idx, d_hitbox_idx);
				o.set_mass(mass_kg);
			}
			else if (obj_type == 2) {
				ObjIndexs obj_idx;

				d_ModelInstance d_mi = create_instance(Runtime::find_host_model_index(vis_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(vis_model_name)->get_triangle_count(), false, 1.0f);
				Runtime::model_instances.push_back(d_mi);
				uint32_t d_mi_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);
				d_ModelInstance d_hitbox = create_instance(Runtime::find_host_model_index(hitbox_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(hitbox_model_name)->get_triangle_count(), true, 1.0f);
				Runtime::model_instances.push_back(d_hitbox);
				uint32_t d_hitbox_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

				obj_idx.player_index = 0;
				o = Object(ObjectType::Player, obj_idx, obj_name, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model_index(vis_model_name), d_mi_idx, d_hitbox_idx);
				o.set_mass(mass_kg);
			}
			else if (obj_type == 3) {
				ObjIndexs obj_idx;

				d_ModelInstance d_mi = create_instance(Runtime::find_host_model_index(vis_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(vis_model_name)->get_triangle_count(), false, 0.001f);
				Runtime::model_instances.push_back(d_mi);
				uint32_t d_mi_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);
				d_ModelInstance d_hitbox = create_instance(Runtime::find_host_model_index(hitbox_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(hitbox_model_name)->get_triangle_count(), true, 0.001f);
				Runtime::model_instances.push_back(d_hitbox);
				uint32_t d_hitbox_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

				obj_idx.weapon_index= Runtime::find_weapon_index("Default Weapon");
				o = Object(ObjectType::Weapon, obj_idx, obj_name, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model_index(vis_model_name), d_mi_idx, d_hitbox_idx);
				o.set_mass(mass_kg);
			}
			else if (obj_type == 4) {
				ObjIndexs obj_idx;

				d_ModelInstance d_mi = create_instance(Runtime::find_host_model_index(vis_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(vis_model_name)->get_triangle_count(), false, 1.0f);
				Runtime::model_instances.push_back(d_mi);
				uint32_t d_mi_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);
				d_ModelInstance d_hitbox = create_instance(Runtime::find_host_model_index(hitbox_model_name), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(hitbox_model_name)->get_triangle_count(), true, 1.0f);
				Runtime::model_instances.push_back(d_hitbox);
				uint32_t d_hitbox_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

				obj_idx.physics_object_index = Runtime::OBJECTS.size() - 1;
				o = Object(ObjectType::Static, obj_idx, obj_name, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model_index(vis_model_name), d_mi_idx, d_hitbox_idx);
				o.set_mass(mass_kg);
			}
			Runtime::OBJECTS.push_back(o);

			if (o.get_object_type() == ObjectType::Player) {
				Runtime::PLAYER_OBJECT = &Runtime::OBJECTS[Runtime::OBJECTS.size() - 1];
			}
	}
	catch (...) {
		// abort select on failure, don't let exception propogate thru sqlite3 call-stack
		return 1;
	}
	return 0;
}

void Runtime::load_objects(sqlite3* sql) {
	const char* command = "SELECT * FROM objects;";
	char* err;

	std::cout << "Loading Models" << std::endl << std::endl;

	std::cout << "	Loading Objects" << std::endl << std::endl;

	Runtime::OBJECTS = *new std::vector<Object>();

	int r = sqlite3_exec(sql, command, sql_callback_object, NULL, &err);
	if (r != SQLITE_OK) {
		std::cerr << "SQL Objects error: " << err << std::endl;
		return;
	}
}

Object* find_object(std::string n) {
	uint32_t c = 0;
	for (Object o : Runtime::OBJECTS) {
		if (o.get_name() == n) {
			return Runtime::OBJECTS.data() + (sizeof(Object) * c);
		}
		c++;
	}
}
uint32_t Runtime::find_object_index(std::string n) {
	uint32_t c = 0;
	for (Object o : Runtime::OBJECTS) {
		if (o.get_name() == n) {
			return c;
		}
		c++;
	}
}

uint32_t Runtime::find_weapon_index(std::string n) {
	uint32_t c = 0;
	for (BulletWeapon w : Runtime::WEAPONS) {
		if (w.get_name() == n) {
			return c;
		}
		c++;
	}
}


void Camera::add_to_euler_direction(glm::vec2 rot) {
	float x = rot.x, y = rot.y;
	//printf("X,Y input mouse coord = {%.2f, %.2f}\n", rot.x, rot.y);
	float normalized_coord_x = ((rot.x - (static_cast<float>(this->dims.x) * 0.5f)) / static_cast<float>(this->dims.x));
	float normalized_coord_y = ((rot.y - (static_cast<float>(this->dims.y) * 0.5f)) / static_cast<float>(this->dims.y));
	//printf("X,Y normalized input mouse coord = {%.2f, %.2f}\n", normalized_coord_x, normalized_coord_y);

	float aspect_ratio = static_cast<float>(this->dims.x) / static_cast<float>(this->dims.y);

	float fov_hori_rad = this->fov.x;
	float fov_vert_rad = this->fov.y;
	float half_fov_hori_rad = fov_hori_rad * 0.5f;
	float half_fov_vert_rad = fov_vert_rad * 0.5f;

	float view_x = normalized_coord_x * half_fov_hori_rad * aspect_ratio;
	float view_y = normalized_coord_y * half_fov_vert_rad;

	this->euler_direction.x += view_x * Runtime::X_SENSITIVITY * aspect_ratio; //* (static_cast<float>(this->dims.x) / this->dims.y);
	this->euler_direction.y -= view_y * Runtime::Y_SENSITIVITY;
	this->euler_direction.z = 0.0f;

	if (this->euler_direction.y > 90.0f) {
		this->euler_direction.y = 90.0f;
	}
	if (this->euler_direction.y < -90.0f) {
		this->euler_direction.y = -90.0f;
	}

	float yaw = this->euler_direction.x * (PI / 180.0f),
		pitch = this->euler_direction.y * (PI / 180.0f);

	this->direction.x = cosf(yaw) * cosf(pitch);
	this->direction.y = sinf(pitch);
	this->direction.z = sinf(yaw) * cosf(pitch);

	this->direction = glm::normalize(this->direction);
}

void Camera::forward(float t) {

	float mag = sqrtf(this->direction.x * this->direction.x + this->direction.y * this->direction.y + this->direction.z + this->direction.z);

	this->position += t * Runtime::BASE_SPEED * this->direction;
}

void Camera::backward(float t) {

	float mag = sqrtf(this->direction.x * this->direction.x + this->direction.y * this->direction.y + this->direction.z + this->direction.z);

	this->position -= t * Runtime::BASE_SPEED * this->direction;
}

void Camera::right(float t) {
	this->position += t * Runtime::BASE_SPEED * glm::normalize(glm::cross(this->direction, glm::vec3(0, 1, 0)));
}

void Camera::left(float t) {
	this->position -= t * Runtime::BASE_SPEED * glm::normalize(glm::cross(this->direction, glm::vec3(0, 1, 0)));
}

int sql_callback_weapon(void* p_data, int num_fields, char** p_fields, char** p_col_names)
{
	try {
		std::cout << "	Weapons:" << std::endl;
		for (size_t h = 0; h < (num_fields / 7) + 1; h++) {
			BulletWeapon w = BulletWeapon();
			if (num_fields - (h * 7) >= 7) {
				for (size_t i = 0; i < 7; i++) {
					char* str = p_fields[h * 7 + i];
					std::istringstream in(str);

					int wpn_type = -1;
					WeaponType type;

					float w_speed = 0.0f;
					float r_speed = 0.0f;
					float b_dmg = 0.0f;
					float f_delay = 0.01f;
					std::string model_name;

					uint32_t model_index = 0;
					d_ModelInstance d_m;
					uint32_t inst_idx = 0;

					switch (i) {
					case 0:
						w.set_name(str);
						break;
					case 1:
						in >> wpn_type;

						type = WeaponType::SemiAutomatic;
						if (wpn_type == 0) {
							type = WeaponType::SemiAutomatic;
						}
						else if (wpn_type == 1) {
							type = WeaponType::FullyAutomatic;
						}
						else if (wpn_type == 2) {
							type = WeaponType::Burst;
						}
						else if (wpn_type == 3) {
							type = WeaponType::BoltAction;
						}
						else if (wpn_type == 4) {
							type = WeaponType::SingleShot;
						}
						else if (wpn_type == 5) {
							type = WeaponType::Melee;
						}

						w.set_weapon_type(type);

						break;
					case 2:
						in >> w_speed;

						w.set_walk_speed(w_speed);

						break;
					case 3:
						in >> r_speed;

						w.set_run_speed(r_speed);

						break;
					case 4:
						in >> b_dmg;

						w.set_base_damage(b_dmg);

						break;
					case 5:
						in >> f_delay;

						w.set_fire_delay(f_delay);

						break;
					case 6:
						in >> model_name;

						model_index = Runtime::find_host_model_index(model_name);

						d_m = create_instance(model_index, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(model_name)->get_triangle_count(), false, 1.0f);
						Runtime::model_instances.push_back(d_m);

						inst_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

						w.set_instance_index(inst_idx);
						break;

					}
					std::cout << str << std::endl;
				}
			}
			else if (num_fields - (h * 7) < 7) {
					for (size_t i = 0; i < num_fields - (h * 7); i++) {
						char* str = p_fields[h * 7 + i];
						std::istringstream in(str);

						int wpn_type = -1;
						WeaponType type;

						float w_speed = 0.0f;
						float r_speed = 0.0f;
						float b_dmg = 0.0f;
						float f_delay = 0.01f;
						std::string model_name;

						uint32_t model_index = 0;
						d_ModelInstance d_m;
						uint32_t inst_idx = 0;

						switch (i) {
						case 0:
							w.set_name(str);
							break;
						case 1:
							in >> wpn_type;

							type = WeaponType::SemiAutomatic;
							if (wpn_type == 0) {
								type = WeaponType::SemiAutomatic;
							}
							else if (wpn_type == 1) {
								type = WeaponType::FullyAutomatic;
							}
							else if (wpn_type == 2) {
								type = WeaponType::Burst;
							}
							else if (wpn_type == 3) {
								type = WeaponType::BoltAction;
							}
							else if (wpn_type == 4) {
								type = WeaponType::SingleShot;
							}
							else if (wpn_type == 5) {
								type = WeaponType::Melee;
							}

							w.set_weapon_type(type);

							break;
						case 2:
							in >> w_speed;

							w.set_walk_speed(w_speed);

							break;
						case 3:
							in >> r_speed;

							w.set_run_speed(r_speed);

							break;
						case 4:
							in >> b_dmg;

							w.set_base_damage(b_dmg);

							break;
						case 5:
							in >> f_delay;

							w.set_fire_delay(f_delay);

							break;
						case 6:
							in >> model_name;

							model_index = Runtime::find_host_model_index(model_name);

							d_m = create_instance(model_index, glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 0.0f, 0.0f), Runtime::find_host_model(model_name)->get_triangle_count(), false, 0.1f);
							Runtime::model_instances.push_back(d_m);

							inst_idx = static_cast<uint32_t>(Runtime::model_instances.size() - 1);

							w.set_instance_index(inst_idx);

						}
						std::cout << str << std::endl;
					}
				}
			w.set_offset(glm::vec3(0.25f, 0.25f, 0.25f));

			Runtime::WEAPONS.push_back(w);
		}
		
	}
	catch (...) {
		// abort select on failure, don't let exception propogate thru sqlite3 call-stack
		return 1;
	}
	return 0;
}

void Runtime::load_weapons(sqlite3* sql) {
	const char* command = "SELECT * FROM weapons;";
	char* err;

	Runtime::WEAPONS = *new std::vector<BulletWeapon>();

	int r = sqlite3_exec(sql, command, sql_callback_weapon, NULL, &err);
	if (r != SQLITE_OK) {
		std::cerr << "SQL Weapons error: " << err << std::endl;
		return;
	}
}

bool Runtime::host_models_contains(std::string name) {
	for (HostModel mod : HOST_MODELS) {
		if (mod.get_name() == name) {
			return true;
		}
	}
	return false;
}

void Level::load_from(std::string path) {

	std::ifstream in;
	in.open(path, std::ios::in);
	if (!in) {
		std::cout << "Cannot find Level: " << path << std::endl;
		return;
	}

	std::cout << "Loading Level: " << path << std::endl;

	this->objects = *new std::vector<Object>();

	std::string line;
	std::getline(in, line);
	std::istringstream parse(line);
	size_t leng = 0;

	parse >> leng;
	std::cout << leng << " static models detected!" << std::endl;
	std::string model;

	thrust::host_vector<d_Model> d_models(DEVICE_MODELS.size());
	uint32_t model_count = 0;

	for (size_t i = 0; i < leng; i++) {
		std::getline(in, line);
		std::istringstream in0(line);

		float x, y, z, x_r, y_r, z_r;

		in0 >> x >> y >> z >> x_r >> y_r >> z_r >> model;
		//std::cout << model << std::endl;

		glm::vec3 position = glm::vec3(x, y, z);
		glm::vec3 rotation = glm::vec3(x_r, y_r, z_r);

		HostModel* h_model = Runtime::find_host_model(model);

		Runtime::model_instances.push_back(create_instance(Runtime::find_host_model_index(model), position, rotation, Runtime::find_host_model(model)->get_triangle_count(), false, 1.0f));

		std::cout << "d_model = " << Runtime::model_instances.back().model_index << std::endl;

		//d_Model d_model2 = Runtime::find_host_model(model)->to_gpu();

		//d_models.push_back(DEVICE_MODELS.at(Runtime::find_host_model_index(model)));	// -> Dis piece o shite

	}
	thrust::copy(DEVICE_MODELS.begin(), DEVICE_MODELS.end(), d_models.begin());
	size_t light_leng;
	std::string line2;
	std::getline(in, line2);
	std::istringstream parse2(line2);
	parse2 >> light_leng;
	std::cout << light_leng << " lights detected!" << std::endl;
	std::vector<d_PointLight> point_lights;
	for (size_t i = 0; i < light_leng; i++) {
		std::getline(in, line);
		std::istringstream in1(line);

		float x, y, z, r, g, b, a, s_r, s_g, s_b, s_a, intensity, falloff, range;

		in1 >> x >> y >> z >> r >> g >> b >> a >> s_r >> s_g >> s_b >> s_a >> intensity >> falloff >> range;
		//std::cout << model << std::endl;

		glm::vec3 position = glm::vec3(x, y, z);
		glm::vec4 color = glm::vec4(r, g, b, a);
		glm::vec4 s_color = glm::vec4(s_r, s_g, s_b, s_a);

		point_lights.push_back(d_PointLight{ position, color, s_color, intensity, falloff, range });
	}

	std::getline(in, line);
	std::istringstream in2(line);

	float r, g, b, a, s_r, s_g, s_b, s_a, intensity;

	in2 >> r >> g >> b >> a >> s_r >> s_g >> s_b >> s_a >> intensity;

	d_AmbientLight amb_light = { glm::vec4(r, g, b, a), glm::vec4(s_r, s_g, s_b, s_a), intensity };

	this->d_model_instance_count = static_cast<uint32_t>(Runtime::model_instances.size());

	this->d_point_lights_count = static_cast<uint32_t>(point_lights.size());

	std::getline(in, line);
	std::istringstream in3(line);
	size_t object_count;
	in3 >> object_count;
	std::cout << std::setw(10) << object_count << " objects detected!" << std::endl;
	bool has_player = false;
	this->objects = *new std::vector<Object>();
	for (size_t i = 0; i < object_count; i++) {
		std::getline(in, line);
		std::istringstream in_obj(line);
		uint32_t type = 0;
		float x, y, z, x_d, y_d, z_d;
		std::string visual_model, rigid_model;

		in_obj >> x >> y >> z >> x_d >> y_d >> z_d >> visual_model;

		ObjectType obj_type = ObjectType::AI;
		//std::cout << "Type " << type << std::endl;

		d_ModelInstance instance;
		uint32_t model_idx = 0, instance_idx = 0, hitbox_index = 0;

		this->objects.push_back(Runtime::OBJECTS[Runtime::find_object_index(visual_model)]);
		this->objects.back().set_position(glm::vec3(x, y, z));
		this->objects.back().set_direction(glm::vec3(x_d, y_d, z_d));

		this->add_model_instance(create_instance(this->objects.back().get_model_index(), this->objects.back().get_position(), this->objects.back().get_direction(), HOST_MODELS[this->objects.back().get_model_index()].get_triangle_count(), false, 1.0f));
		this->objects.back().set_instance_index(static_cast<uint32_t>(Runtime::model_instances.size() - 1));
		this->objects.back().set_spawn_point(glm::vec3(x, y, z));

		std::cout << "Object added: " << &this->objects.back() << " @ " << this->objects.back().get_model_index() << " index with model " << visual_model << " of type " << this->objects.back().get_object_type() << std::endl;
	}

	error_check(hipMalloc((void**)&this->d_DEVICE_MODELS, sizeof(d_Model) * d_models.size()));
	error_check(hipMemcpy(this->d_DEVICE_MODELS, thrust::raw_pointer_cast(d_models.data()), sizeof(d_Model) * d_models.size(), hipMemcpyHostToDevice));
	this->d_DEVICE_MODEL_COUNT = static_cast<uint32_t>(d_models.size());

	error_check(hipMalloc((void**)&this->d_ambient_light, sizeof(d_AmbientLight)));
	error_check(hipMemcpy(this->d_ambient_light, &amb_light, sizeof(d_AmbientLight), hipMemcpyHostToDevice));

	error_check(hipMalloc((void**)&this->d_point_lights, sizeof(d_PointLight) * point_lights.size()));
	error_check(hipMemcpy(this->d_point_lights, point_lights.data(), sizeof(d_PointLight) * point_lights.size(), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	//this->upload_objects();

	//error_check(hipMalloc((void**)&this->d_model_instances, sizeof(d_ModelInstance) * this->model_instances.size()));
	//this->upload_instances();
}

static void keyboard_callback(GLFWwindow* win, int key, int scancode, int action, int mods) {
	Runtime::KEY_USED = true;

	//std::cout << "Key Calledback!" << std::endl;

	Runtime::CURRENT_KEY = key;
	Runtime::CURRENT_SCANCODE = scancode;
	Runtime::CURRENT_ACTION = action;
	Runtime::CURRENT_MODS = mods;
}

static void mouse_callback(GLFWwindow* window, int button, int action, int mods) {
	Runtime::MOUSE_USED = true;

	Runtime::CURRENT_MOUSE = button;
	Runtime::CURRENT_ACTION = action;
	Runtime::CURRENT_MODS = mods;
}

Application::Application(int32_t dimx, int32_t dimy) {
	this->dims = glm::ivec2(dimx, dimy);

	int err = sqlite3_open("databases\\master.db", &this->database_connection);
	if (err != SQLITE_OK) {
		std::cout << "Cannot open asset database! : " << sqlite3_errmsg(this->database_connection) << std::endl;
		return;
	}
	Runtime::model_instances = *new std::vector<d_ModelInstance>();
	Runtime::runtime_load(this->database_connection);
	Runtime::load_objects(this->database_connection);
	Runtime::load_weapons(this->database_connection);

	glfwInit();
	glfwSwapInterval(0);
	glfwWindowHint(GLFW_CENTER_CURSOR, GLFW_TRUE);
	glfwWindowHint(GLFW_VISIBLE, GLFW_TRUE);
	glfwWindowHint(GLFW_RESIZABLE, GLFW_TRUE);

	this->win = glfwCreateWindow(this->dims.x, this->dims.y, "ZDproto-demo v0.00", NULL, NULL);
	this->loop = true;

	glm::vec3 init_position = glm::vec3(10.0f, 10.0f, 0.0f);
	glm::vec3 init_direction = glm::vec3(0.0f, 0.0f, 0.0f);

	this->camera = new Camera(this->dims, 120.0f, init_position, init_direction);

	this->level = new Level("resources/levels/test_level.txt", this->camera);

	//this->ai.push_back(FiniteStateMachine(glm::vec3(10.0f, 0.0f, 0.0f), this->level, ));


	glfwMakeContextCurrent(this->win);
}

void Application::input_handle() {
	std::cout << Runtime::CURRENT_KEY << std::endl;
	switch (Runtime::CURRENT_KEY) {
	case GLFW_KEY_W:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT || Runtime::CURRENT_ACTION == GLFW_RELEASE) {
			this->camera->forward(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_S:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->backward(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_A:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->left(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_D:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->right(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_R:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->cam->set_capture_mode(RT);
		}
		break;
	case GLFW_KEY_F:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->camera->set_capture_mode(FULLBRIGHT);
		}
		break;
	case GLFW_KEY_ESCAPE:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->loop = false;
		}
		break;
	case GLFW_KEY_P:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->cam->get_rays(0)->debug_stats();
		}
		break;
	case GLFW_MOUSE_BUTTON_LEFT:
		if (Runtime::CURRENT_ACTION == GLFW_REPEAT) {

		}
		break;
	}
}

void Application::mouse_handle() {

	switch (Runtime::CURRENT_MOUSE) {
	case GLFW_MOUSE_BUTTON_LEFT:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			glm::vec3 fire_direction = this->camera->get_direction();

			BulletWeapon* current_weapon = Runtime::PLAYER_OBJECT->get_current_weapon();
			bool fire = false;
			switch (current_weapon->get_weapon_type()) {
				case WeaponType::SemiAutomatic:
					if (glfwGetTime() - this->camera->get_last_time() >= current_weapon->get_fire_delay()) {
						fire = true;
					}
					break;
				case WeaponType::FullyAutomatic:
					if (glfwGetTime() - this->camera->get_last_time() >= current_weapon->get_fire_delay()) {
						fire = true;
					}
					break;
			}

			if (fire) {

				std::cout << "Shooting!" << std::endl;

				bool* intersection_tests = new bool[this->level->get_object_count()], * d_intersection_tests;
				float* distances = new float[this->level->get_object_count()], * d_distances;
				error_check(hipMalloc((void**)&d_intersection_tests, this->level->get_object_count() * sizeof(bool)), "Application::mouse_handle cudaMalloc1");
				error_check(hipMalloc((void**)&d_distances, this->level->get_object_count() * sizeof(float)), "Application::mouse_handle cudaMalloc2");
				//std::cout << this->level->get_d_object_count() << " Objects going in @ " << this->level->get_d_objects() << std::endl;
				test_intersection << <(this->level->get_d_object_count() / 128) + 1, 128 >> > (this->camera->get_position(), fire_direction, this->level->get_d_objects(), this->level->get_d_object_count(), this->level->get_d_model_instances(), this->level->get_d_model_instance_count(), this->level->get_d_device_models(), d_intersection_tests, d_distances);
				hipDeviceSynchronize();
				error_check(hipGetLastError(), "Application::mouse_handle kernel call");

				error_check(hipMemcpy(intersection_tests, d_intersection_tests, this->level->get_object_count() * sizeof(bool), hipMemcpyDeviceToHost), "Application::mouse_handle cudaMemcpy1");
				error_check(hipMemcpy(distances, d_distances, this->level->get_object_count() * sizeof(float), hipMemcpyDeviceToHost), "Application::mouse_handle cudaMemcpy2");
				float closest = 1000.0f;
				int32_t index = -1;
				for (size_t i = 0; i < this->level->get_object_count(); i++) {
					if (intersection_tests[i]) {
						if (distances[i] < closest) {
							index = static_cast<int32_t>(i);
						}
					}
				}
				if (index != -1) {
					std::cout << "Damaging object " << index << std::endl;
					Object objs = this->level->get_objects_ptr()[index];
					objs.set_health(objs.get_health() - 25.0f);
					this->level->update_object(static_cast<uint32_t>(index), objs);
				}

				error_check(hipFree(d_intersection_tests));
				error_check(hipFree(d_distances));
			}
		}
		break;

	}
}

void Application::main_loop() {
	this->camera->set_last_time(glfwGetTime());
	glfwSetKeyCallback(this->win, keyboard_callback);
	glfwSetMouseButtonCallback(this->win, mouse_callback);
	glfwMakeContextCurrent(this->win);

	//this->camera->set_position(Runtime::PLAYER_OBJECT->get_position());

	Runtime::PLAYER_OBJECT->set_primary_weapon(&Runtime::WEAPONS[Runtime::find_weapon_index("Default Weapon")]);

	int frame_count = 0;
	while (this->loop && !glfwWindowShouldClose(this->win)) {

		glfwPollEvents();
		this->frame_buffer = new glm::vec4[this->dims.x * this->dims.y];
		glm::vec4* d_frame_buffer;

		error_check(hipMalloc((void**)&d_frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x));

		this->camera->new_frame();

		this->zero_frame_buffer_sse();

		error_check(hipMemcpy(d_frame_buffer, this->frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x, hipMemcpyHostToDevice));

		Object* obj = this->level->get_objects_ptr();
		//std::cout << "Updating " << this->level->get_object_count() << " objects in world!" << std::endl;
		for (size_t i = 0; i < this->level->get_object_count(); i++) {
			obj[i].update(&Runtime::model_instances[obj[i].get_instance_index()], &Runtime::model_instances[obj[i].get_hitbox_instance_index()], this->camera, glfwGetTime() - this->camera->get_last_time(), this->win, Runtime::PLAYER_OBJECT);
			/*if (obj[i].get_object_type() == ObjectType::Player && obj[i].get_health() <= 0.0f) {
				std::cout << "Game Over" << std::endl;
				this->loop = false;
			}*/
		}

		if (frame_count != 0) {
			level->clean_d_objects();
		}

		this->level->upload_objects();
		this->level->upload_instances();

		if (Runtime::KEY_USED) {
			this->input_handle();
			Runtime::control::reset_key();
		}
		if (Runtime::MOUSE_USED) {
			this->mouse_handle();
			Runtime::control::reset_mouse();
		}

		// Render functions
		this->camera->capture(this->level->get_d_model_instances(), this->level->get_d_model_instance_count(), this->level->get_d_device_models(), this->level->get_d_ambient_light(), this->level->get_d_point_lights(), this->level->get_d_point_lights_size(), d_frame_buffer);

		//this->camera->copy_to_frame_buffer(this->frame_buffer, 0);

		error_check(hipMemcpy(this->frame_buffer, d_frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x, hipMemcpyDeviceToHost));


		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glDrawPixels(this->dims.x, this->dims.y, GL_BGRA_EXT, GL_FLOAT, this->frame_buffer);
		glfwSwapBuffers(this->win);

		this->camera->cleanup_frame();

		delete this->frame_buffer;
		hipFree(d_frame_buffer);

		this->camera->debug_print();
		//this->cam->last_time = glfwGetTime();
		this->camera->set_last_time(glfwGetTime());
		frame_count++;
	}
	glfwDestroyWindow(this->win);
	glfwTerminate();
}

void Level::add_model_instance(d_ModelInstance inst) {
	Runtime::model_instances.push_back(inst);
}

void Level::update_instance(uint32_t index, d_ModelInstance model) {
	Runtime::model_instances[index] = model;
}

void Level::update_object(uint32_t index, Object object) {
	this->objects[index] = object;

	d_ModelInstance instance = Runtime::model_instances[this->objects[index].get_instance_index()];
	instance.position = this->objects[index].get_position();
	instance.rotation = this->objects[index].get_direction();
	this->update_instance(this->objects[index].get_instance_index(), instance);
}

void Level::upload_instances() {
	error_check(hipMalloc((void**)&this->d_model_instances, sizeof(d_ModelInstance) * Runtime::model_instances.size()));
	error_check(hipMemcpy(this->d_model_instances, Runtime::model_instances.data(), sizeof(d_ModelInstance) * Runtime::model_instances.size(), hipMemcpyHostToDevice));
	hipDeviceSynchronize();
	this->d_model_instance_count = static_cast<uint32_t>(Runtime::model_instances.size());
}

Object::Object(ObjectType type, ObjIndexs idxs, std::string name, glm::vec3 position, glm::vec3 direction, uint32_t model, uint32_t instance, uint32_t hitbox_instance) {
	this->name = name;
	this->position = position;
	this->spawn_point = position;
	this->direction = direction;
	this->object_type = type;
	this->rotation = glm::vec3(0.0f, 0.0f, 0.0f);

	this->model_index = model;
	this->instance_index = instance;
	this->hitbox_instance_index = hitbox_instance;

	this->obj_indices = idxs;

	this->creation_time = glfwGetTime();

	this->primary = &Runtime::WEAPONS[Runtime::find_weapon_index("Default Weapon")];
}

void Object::update(d_ModelInstance* instances, d_ModelInstance* hitbox_instance, Camera* cam, float t, GLFWwindow* win, Object* player) {
	//std::cout << "updating object" << std::endl;
	float to_player, time;
	if (this->object_type == ObjectType::AI) {
		//std::cout << "	updating as AI" << std::endl;
		this->target_position = player->get_position();
		this->direction = glm::normalize(this->target_position - this->position);
		this->position += this->direction * t * 5.0f;
		instances->position = this->position;
		instances->rotation = this->direction;

		glm::vec3 dist = this->target_position - this->position;
		to_player = dist.length();
		printf("Distance to player = %.2f, %.2f, %.2f\n", dist.x, dist.y, dist.z);

		hitbox_instance->position = this->position;
		hitbox_instance->rotation = this->direction;

		time = glfwGetTime();
		if (to_player <= this->attack_range) {
			std::cout << std::setw(10) << "AI Within Attack Range" << std::endl;
			if (time - this->last_attack >= this->attack_cooldown) {
				player->set_health(player->get_health() - this->current_damage);
				this->last_attack = time;
				std::cout << "AI Attacking Player" << std::endl;
			}
		}
		if (this->current_health <= 0.0f) {
			this->position = this->spawn_point;
			this->current_health = max_health;
		}
	}

	else if (this->object_type == ObjectType::Player) {
		//std::cout << "	updating as Player" << std::endl;
		double x, y;
		glfwGetCursorPos(win, &x, &y);
		cam->add_to_euler_direction(glm::vec2(static_cast<float>(x), static_cast<float>(y)));
		glfwSetCursorPos(win, cam->get_dims().x * 0.5f, cam->get_dims().y * 0.5f);
		this->direction = cam->get_direction();
		this->position = cam->get_position();
		Runtime::PLAYER_OBJECT = this;

		hitbox_instance->position = this->position;
		hitbox_instance->rotation = this->direction;

		Runtime::PLAYER_OBJECT->set_primary_weapon(&Runtime::WEAPONS[0]);

		d_ModelInstance* d_wpn = &Runtime::model_instances[Runtime::PLAYER_OBJECT->get_current_weapon()->get_instance_index()];
		//d_wpn->position = this->position + this->primary->get_offset();
		//d_wpn->rotation = this->direction;
		//instances[this->primary->get_instance_index()].position = this->position + this->primary->get_offset();
	}

	else if (this->object_type == ObjectType::Weapon) {
		//std::cout << "	updating as Weapon" << std::endl;
		uint32_t inst_idx = Runtime::WEAPONS[this->obj_indices.weapon_index].get_instance_index();
		BulletWeapon* wpn = &Runtime::WEAPONS[this->obj_indices.weapon_index];
		d_ModelInstance* d_mi = &Runtime::model_instances[inst_idx];

		//d_mi->position = Runtime::PLAYER_OBJECT->get_position() + Runtime::WEAPONS[this->obj_indices.weapon_index].get_offset();
		//d_mi->rotation = Runtime::PLAYER_OBJECT->get_direction();
	}

	else if (this->object_type == ObjectType::Physics) {
		float delta_time = this->creation_time - time;

		this->position -= glm::vec3(0.0f, -9.81 * delta_time * delta_time, 0.0f);
		instances[this->instance_index].position = this->position;
	}
}

HostModel* Runtime::find_host_model(std::string name) {
	for (uint32_t i = 0; i < HOST_MODELS.size(); i++) {
		if (HOST_MODELS.at(i).get_name() == name) {
			return &HOST_MODELS.at(i);
		}
	}
	return &HOST_MODELS.at(0);	// Def fix this later, don't know what to do with it now
}

uint32_t Runtime::find_host_model_index(std::string name) {
	for (uint32_t i = 0; i < HOST_MODELS.size(); i++) {
		if (HOST_MODELS.at(i).get_name() == name) {
			return i;
		}
	}
	return 0;
}
