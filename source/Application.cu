	// Application.cpp
#include "../include/Application.h"

static void keyboard_callback(GLFWwindow* win, int key, int scancode, int action, int mods) {
	Runtime::KEY_USED = true;

	//std::cout << "Key Calledback!" << std::endl;

	Runtime::CURRENT_KEY = key;
	Runtime::CURRENT_SCANCODE = scancode;
	Runtime::CURRENT_ACTION = action;
	Runtime::CURRENT_MODS = mods;
}

static void mouse_callback(GLFWwindow* window, int button, int action, int mods) {
	Runtime::MOUSE_USED = true;

	Runtime::CURRENT_MOUSE = button;
	Runtime::CURRENT_ACTION = action;
	Runtime::CURRENT_MODS = mods;
}

Application::Application() {

}

Application::Application(int32_t dimx, int32_t dimy) {
	this->dims = glm::ivec2(dimx, dimy);

	glfwInit();
	glfwSwapInterval(0);
	glfwWindowHint(GLFW_CENTER_CURSOR, GLFW_TRUE);
	glfwWindowHint(GLFW_VISIBLE, GLFW_TRUE);
	glfwWindowHint(GLFW_RESIZABLE, GLFW_TRUE);

	this->win = glfwCreateWindow(this->dims.x, this->dims.y, "ZDproto-demo v0.00", NULL, NULL);
	this->loop = true;

	glm::vec3 init_position = glm::vec3(10.0f, 10.0f, 0.0f);
	glm::vec3 init_direction = glm::vec3(0.0f, 0.0f, 0.0f);

	this->camera = new Camera(this->dims, 120.0f, init_position, init_direction);

	this->level = new Level("resources/levels/test_level.txt", this->camera);

	//this->ai.push_back(FiniteStateMachine(glm::vec3(10.0f, 0.0f, 0.0f), this->level, ));

	glfwMakeContextCurrent(this->win);
}

void Application::input_handle() {
	std::cout << Runtime::CURRENT_KEY << std::endl;
	switch (Runtime::CURRENT_KEY) {
	case GLFW_KEY_W:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT || Runtime::CURRENT_ACTION == GLFW_RELEASE) {
			this->camera->forward(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_S:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->backward(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_A:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->left(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_D:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->camera->right(glfwGetTime() - this->camera->get_last_time());
		}
		break;
	case GLFW_KEY_R:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->cam->set_capture_mode(RT);
		}
		break;
	case GLFW_KEY_F:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->camera->set_capture_mode(FULLBRIGHT);
		}
		break;
	case GLFW_KEY_ESCAPE:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			this->loop = false;
		}
		break;
	case GLFW_KEY_P:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			//this->cam->get_rays(0)->debug_stats();
		}
		break;
	case GLFW_MOUSE_BUTTON_LEFT:
		if (Runtime::CURRENT_ACTION == GLFW_REPEAT) {

		}
		break;
	}
}

void Application::mouse_handle() {
	switch (Runtime::CURRENT_MOUSE) {
	case GLFW_MOUSE_BUTTON_LEFT:
		if (Runtime::CURRENT_ACTION == GLFW_PRESS || Runtime::CURRENT_ACTION == GLFW_REPEAT) {
			glm::vec3 fire_direction = this->camera->get_direction();

			bool* intersection_tests = new bool[this->level->get_object_count()],* d_intersection_tests;
			error_check(hipMalloc((void**)&d_intersection_tests, this->level->get_object_count() * sizeof(bool)), "Application::mouse_handle hipMalloc");
			//std::cout << this->level->get_d_object_count() << " Objects going in @ " << this->level->get_d_objects() << std::endl;
			test_intersection << <(this->level->get_d_object_count() / 128) + 1, 128 >> > (this->camera->get_position(), fire_direction, this->level->get_d_objects(), this->level->get_d_object_count(), this->level->get_d_model_instances(), this->level->get_d_model_instance_count(), this->level->get_d_device_models(), d_intersection_tests);
			hipDeviceSynchronize();
			error_check(hipGetLastError(), "Application::mouse_handle kernel call");

			error_check(hipMemcpy(intersection_tests, d_intersection_tests, this->level->get_object_count() * sizeof(bool), hipMemcpyDeviceToHost), "Application::mouse_handle hipMemcpy");
			
			for (size_t i = 0; i < this->level->get_object_count(); i++) {
				if (intersection_tests[i]) {
					Object objs = this->level->get_objects_ptr()[i];
					objs.set_health(objs.get_health() - 25.0f);
					if (objs.get_health() <= 0.0f) {
						objs.set_position(objs.get_spawn_point());
						objs.set_health(50.0f);
					}
					this->level->update_object(static_cast<uint32_t>(i), objs);
				}
			}

			error_check(hipFree(d_intersection_tests));
		}
		break;

	}
}

void Application::zero_frame_buffer_sse() {
	__m128 to_set = _mm_set_ps(0.0f, 0.0f, 0.0f, 0.0f);

	glm::ivec2 dims = this->camera->get_dims();
	for (uint32_t y = 0; y < dims.y; y++) {
		for (uint32_t x = 0; x < dims.x; x++) {
			_mm_store_ps(&this->frame_buffer[y * dims.x + x].x, to_set);
			//this->frame_buffer[y * dims.x + x] = glm::vec4(0.0f);
		}
	}
}

void Application::main_loop() {
	this->camera->set_last_time(glfwGetTime());
	glfwSetKeyCallback(this->win, keyboard_callback);
	glfwSetMouseButtonCallback(this->win, mouse_callback);
	glfwMakeContextCurrent(this->win);

	int frame_count = 0;
	while (this->loop && !glfwWindowShouldClose(this->win)) {

		glfwPollEvents();
		this->frame_buffer = new glm::vec4[this->dims.x * this->dims.y];
		glm::vec4* d_frame_buffer;

		error_check(hipMalloc((void**)&d_frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x));

		this->camera->new_frame();

		this->zero_frame_buffer_sse();

		error_check(hipMemcpy(d_frame_buffer, this->frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x, hipMemcpyHostToDevice));

		Object* obj = this->level->get_objects_ptr();
		std::cout << "Updating " << this->level->get_object_count() << " objects in world!" << std::endl;
		for (size_t i = 0; i < this->level->get_object_count(); i++) {
			obj[i].update(&this->level->get_model_instances()[obj[i].get_instance_index()], this->camera, glfwGetTime() - this->camera->get_last_time(), this->win);
		}

		if (frame_count != 0) {
			level->clean_d_objects();
		}

		this->level->upload_objects();
		this->level->upload_instances();
		hipDeviceSynchronize();

		if (Runtime::KEY_USED) {
			this->input_handle();
			Runtime::control::reset_key();
		}
		if (Runtime::MOUSE_USED) {
			this->mouse_handle();
			Runtime::control::reset_mouse();
		}

			// Render functions
		this->camera->capture(this->level->get_d_model_instances(), this->level->get_d_model_instance_count(), this->level->get_d_device_models(), this->level->get_d_ambient_light(), this->level->get_d_point_lights(), this->level->get_d_point_lights_size(), d_frame_buffer);

		//this->camera->copy_to_frame_buffer(this->frame_buffer, 0);

		error_check(hipMemcpy(this->frame_buffer, d_frame_buffer, sizeof(glm::vec4) * this->dims.y * this->dims.x, hipMemcpyDeviceToHost));


		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glDrawPixels(this->dims.x, this->dims.y, GL_BGRA_EXT, GL_FLOAT, this->frame_buffer);
		glfwSwapBuffers(this->win);

		this->camera->cleanup_frame();

		delete this->frame_buffer;
		hipFree(d_frame_buffer);

		this->camera->debug_print();
		//this->cam->last_time = glfwGetTime();
		this->camera->set_last_time(glfwGetTime());
		frame_count++;
	}
	glfwDestroyWindow(this->win);
	glfwTerminate();
}