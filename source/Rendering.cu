#include "hip/hip_runtime.h"
	// Rendering.cu
#include "../include/Rendering.cuh"
#include "../include/Camera.h"
#include "../include/Runtime.h"

__device__
glm::vec4 interpolateColor3D(const glm::vec4& c1, const glm::vec4& c2, const glm::vec4& c3,
	float alpha, float beta, float gamma) {
	float r = alpha * c1.x + beta * c2.x + gamma * c3.x;
	float g = alpha * c1.y + beta * c2.y + gamma * c3.y;
	float b = alpha * c1.z + beta * c2.z + gamma * c3.z;
	return glm::vec4(r, g, b, 1.0f);
}

__device__
glm::vec3 calculateBarycentric(const glm::vec2& point, const glm::vec2& vertex0, const glm::vec2& vertex1, const glm::vec2& vertex2) {
	glm::vec2 v0 = vertex1 - vertex0;
	glm::vec2 v1 = vertex2 - vertex0;
	glm::vec2 v2 = point - vertex0;

	float dot00 = glm::dot(v0, v0);
	float dot01 = glm::dot(v0, v1);
	float dot02 = glm::dot(v0, v2);
	float dot11 = glm::dot(v1, v1);
	float dot12 = glm::dot(v1, v2);

	float denom = dot00 * dot11 - dot01 * dot01;
	float barycentricY = (dot11 * dot02 - dot01 * dot12) / denom;
	float barycentricZ = (dot00 * dot12 - dot01 * dot02) / denom;
	float barycentricX = 1.0f - barycentricY - barycentricZ;

	return glm::vec3(barycentricX, barycentricY, barycentricZ);
}

__device__
bool ray_intersects_box(glm::vec3 position, glm::vec3 direction, glm::vec3& box_min, glm::vec3& box_max, int& towards) {

	glm::vec3 invDirection = 1.0f / direction;

	// Calculate intersection distances
	glm::vec3 t1 = (box_min - position) * invDirection;
	glm::vec3 t2 = (box_max - position) * invDirection;

	// Find the maximum and minimum of these intersection distances
	glm::vec3 tMin = glm::min(t1, t2);
	glm::vec3 tMax = glm::max(t1, t2);

	float tNear = glm::max(glm::max(tMin.x, tMin.y), tMin.z);
	float tFar = glm::min(glm::min(tMax.x, tMax.y), tMax.z);

	if (tFar >= tNear) {
		if (tNear >= 0.0f) {
			towards = 1;
		}
		else {
			towards = -1;
		}
	}
	else {
		towards = 0;
	}
	
	return tFar >= tNear;
}

__device__
glm::vec4 sample_texture(glm::vec4* texture, glm::ivec2 dims, float x, float y) {
	if (y >= 0.0f && x >= 0.0f && x <= 1.0f && y <= 1.0f) {
		return texture[static_cast<int>(y * dims.y) * dims.x + static_cast<int>(x * dims.x)];
	}
	return glm::vec4(0.0f);
}

void Camera::capture(d_ModelInstance* instances, uint32_t instance_count, d_Model* models, d_Material* materials, d_AmbientLight* amb_light, d_PointLight* point_lights, uint32_t point_lights_size, glm::vec4* buffer, glm::vec4* buffer_post) {

	setup_rays << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->position, this->direction, this->current_fov.x, this->d_ray_matrix[0], this->dims);
	set_visible_tris << < (instance_count / 128) + 1, 128 >> > (this->position, this->direction, this->current_fov, models, instances, instance_count);
	hipDeviceSynchronize();
	for (uint8_t i = 0; i < MAX_BOUNCE_COUNT; i++) {
		capture_with_rays << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->position, this->direction, this->current_fov.x, instances, instance_count, this->d_ray_matrix[i], this->dims, models);
		hipDeviceSynchronize();
	}
	calculate_lighting << < (this->dims.y * this->dims.x) / 128, 128 >> > (amb_light, point_lights, materials, point_lights_size, this->d_ray_matrix[0], this->dims, buffer);
	hipDeviceSynchronize();

	//fxaa_pass << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->dims, buffer, buffer_post);
	//error_check(hipGetLastError(), "fxaa pass");
	//hipDeviceSynchronize();

	//texture_map << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->d_ray_matrix[0], this->dims, buffer);

	//copy_frame_buffer << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->dims, buffer, fin_buffer);
	//error_check(hipGetLastError(), "copy_frame_buffer");


	if (Runtime::PLAYER_OBJECT->get_player_state() == PlayerState::RunF) {
		draw_crosshair << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->dims, buffer, Runtime::PLAYER_OBJECT->get_current_weapon()->get_crosshair(), true, false);
		error_check(hipGetLastError(), "draw_crosshair RUNNING");
	}
	else {
		draw_crosshair << < (this->dims.y * this->dims.x) / 128, 128 >> > (this->dims, buffer, Runtime::PLAYER_OBJECT->get_current_weapon()->get_crosshair(), false, true);
		error_check(hipGetLastError(), "draw_crosshair IDLE");
	}
	hipDeviceSynchronize();
}

__global__
void setup_rays(glm::vec3 position, glm::vec3 direction, float horizontal_fov, Ray* rays, glm::ivec2 dims) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	uint32_t idx = y * dims.x + x;

	if (!(x >= dims.x && x < 0) && !(y >= dims.y && y < 0)) {
		Ray* ray = &rays[idx];

		//ray->position = position;

		float ratio = static_cast<float>(dims.x) / static_cast<float>(dims.y);
		float norm_x = (x - (static_cast<float>(dims.x) * 0.5f)) / (static_cast<float>(dims.x) * 0.5f);
		float norm_y = (y - (static_cast<float>(dims.y) * 0.5f)) / (static_cast<float>(dims.y) * 0.5f);
		float fov_rad = horizontal_fov * (PI / 180.0f);
		float half_fov = fov_rad * 0.5f;

		glm::vec3 right = glm::cross(direction, glm::vec3(0.0f, 1.0f, 0.0f));

		right = glm::normalize(right);

		glm::vec3 up = glm::cross(right, direction);

		up = glm::normalize(up);

		ray->direction = direction + norm_x * half_fov * ratio * right + norm_y * half_fov * up;
	}
}

__global__
void set_visible_tris(glm::vec3 position, glm::vec3 direction, glm::vec2 fov, d_Model* models, d_ModelInstance* instances, uint32_t instance_count) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i);
	uint32_t idx = x;

	if (x < instance_count) {
		if (!instances[idx].is_hitbox) {
			glm::vec3 min_direction_x = glm::vec3((((-fov.x * 0.5f) / fov.x) * (PI / 180.0f)) * direction.x, direction.y, direction.z);
			glm::vec3 max_direction_x = glm::vec3((((fov.x * 0.5f) / fov.x) * (PI / 180.0f)) * direction.x, direction.y, direction.z);
			glm::vec3 min_direction_y = glm::vec3(direction.x, (((-fov.y * 0.5f) / fov.y) * (PI / 180.0f)) * direction.y, direction.z);
			glm::vec3 max_direction_y = glm::vec3(direction.x, (((-fov.y * 0.5f) / fov.y) * (PI / 180.0f)) * direction.y, direction.z);

			d_ModelInstance* instance = &instances[idx];
			d_Model* model = &models[instance->model_index];
			for (uint32_t k = 0; k < *model->triangle_count; k++) {

				float answers[5];
				answers[0] = glm::dot(model->triangles[k].normal, min_direction_x);
				answers[1] = glm::dot(model->triangles[k].normal, max_direction_x);
				answers[2] = glm::dot(model->triangles[k].normal, min_direction_y);
				answers[3] = glm::dot(model->triangles[k].normal, max_direction_y);
				answers[4] = glm::dot(model->triangles[k].normal, direction);

				bool vis = false;
				for (uint8_t m = 0; m < 5; m++) {
					if (answers[m] <= 0.0f) {
						vis = true;
						break;
					}
				}

				if (vis) {
					instance->visible_triangles[k] = true;
				}
				else {
					instance->visible_triangles[k] = false;
				}
				instance->visible_triangles[k] = true;
			}
		}
		else {
			return;
		}
	}
	else {
		return;
	}
}

__global__
void capture_with_rays(glm::vec3 position, glm::vec3 direction, float horizontal_fov, d_ModelInstance* instances, uint32_t instance_count, Ray* rays, glm::ivec2 dims, d_Model* models) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	uint32_t idx = y * dims.x + x;

	if (!(x >= dims.x || x < 0) && !(y >= dims.y || y < 0)) {
		//printf("ray index = %i\n", idx);
		Ray* ray = &rays[idx];

		ray->position = position;
		bool intersected = false, tried = false;
		float last_leng = 1000.0f;
		//printf("%i # Instances\n", instance_count);
		uint32_t history[16];
		uint32_t history_idx = 0;
		for (int j1 = 0; j1 < instance_count; j1++) {
			d_ModelInstance* g = &instances[j1];
			if (!g->is_hitbox) {
				float scale = g->scale;
				//printf("ModelIndex = %i\n", g->model_index);
				//printf("Model Index %i\n", g->model_index);
				uint32_t c = *(models[g->model_index].triangle_count);
				d_Model* model = &models[g->model_index];
				uint32_t next = model->bvh.initial;;
				bool went_back_and_checked = false;
				BVHNode* node = &model->bvh.nodes[model->bvh.initial];
				for (uint32_t n = 0; n < models[g->model_index].bvh.layers; n++) {
					bool cont = false;
					if (n == 0 && !model->bvh.nodes[model->bvh.initial].volume.is_base && model->bvh.node_size > 1) {
						//printf("BVH initial = %i for size %i\n", model->bvh.initial, model->bvh.node_size);
						glm::vec3 min = node->volume.vertices[0] + g->position, max = node->volume.vertices[1] + g->position;
						int towards = 0;
						bool vol_intersect = ray_intersects_box(ray->position, ray->direction, min, max, towards);

						if (vol_intersect) {
							cont = true;
						}
						/*if (vol_intersect && !node->volume.is_base) {
							if (towards < 0) {
								next = model->bvh.nodes[model->bvh.initial].left_child_index;
								goto contin;
								//printf("Going left to %i\n", next);
							}
							else if (towards > 0) {
								next = model->bvh.nodes[model->bvh.initial].right_child_index;
								goto contin;
								//printf("Going right to %i\n", next);
							}
						}
						else if (node->volume.is_base) {
							cont = true;

						}*/
					}
					else if (model->bvh.nodes[model->bvh.initial].volume.is_base){
						cont = true;
					}

					if (cont) {
						//printf("Next Node = %i out of %i nodes\n", next, model->bvh.node_size);
						//BVHNode* node = &model->bvh.nodes[next];

						bool intersect_test = false;
						if (!node->volume.is_base) {
							//printf("Left idx = %i, Right idx = %i\n", node->left_child_index, node->right_child_index);
							BVHNode* left = &model->bvh.nodes[node->left_child_index],
								* right = &model->bvh.nodes[node->right_child_index];
							glm::vec3 min = left->volume.vertices[0] + g->position, max = left->volume.vertices[1] + g->position;
							glm::vec3 min_right = right->volume.vertices[0] + g->position, max_right = right->volume.vertices[1] + g->position;
							int towards = 0, towards_right = 0;
							bool vol_intersect_a = ray_intersects_box(ray->position, ray->direction, min, max, towards);
							bool vol_intersect_b = ray_intersects_box(ray->position, ray->direction, min_right, max_right, towards_right);
							bool corrected = false;
							if (vol_intersect_a && vol_intersect_b) {

								if (towards > 0) {
									vol_intersect_a = true;
									vol_intersect_b = false;
									next = left->right_child_index;
									corrected = true;
								}
								else if (towards < 0) {
									vol_intersect_a = true;
									vol_intersect_b = false;
									next = left->left_child_index;
									corrected = true;
								}
								if (towards_right > 0) {
									vol_intersect_a = false;
									vol_intersect_b = true;
									next = right->right_child_index;
									corrected = true;
								}
								else if (towards_right < 0) {
									vol_intersect_a = false;
									vol_intersect_b = true;
									next = right->left_child_index;
									corrected = true;
								}

								if (corrected) {
									n++;
								}
							}

							if (vol_intersect_a && !corrected) {
								next = node->left_child_index;
							}
							else if (vol_intersect_b && !corrected) {
								next = node->right_child_index;
							}
							if (corrected) {
								continue;
							}
							if (!vol_intersect_a && !vol_intersect_b) {
								/*if (next == node->left_child_index) {
									next = node->right_child_index;
								}
								else if (next == node->right_child_index) {
									next = node->left_child_index;
								}*/
								//printf("BVH going nowhere\n");
								goto contin;
								//printf("BVH going nowhere\n");
								//continue;
							}
						}
						if (node->volume.is_base) {
							//printf("Node is base w/ %i triangles!\n", node->volume.triangle_count);

							for (uint32_t p = 0; p < static_cast<uint32_t>(node->volume.triangle_count); p++) {
								tried = true;
								//printf("Checking Triangle %i\n", node->volume.triangles[p]);
								//printf("Before Grabbing Triangle %i!\n", node->volume.triangles[p]);
								Tri* t = &model->triangles[node->volume.triangles[p]];
								//printf("After grabbing triangle!\n");
								Vertex* vs = model->vertices;
								glm::vec3 offset = instances[j1].position;
								glm::vec3 dir = instances[j1].rotation;

								glm::vec2 intersection;
								float d;
								glm::vec2 uv;

								glm::vec3 rotation_axis = glm::vec3(0.0f, 0.0f, 1.0f);
								float angle = glm::acos(glm::dot(rotation_axis, rotation_axis));
								glm::vec3 axis = glm::cross(rotation_axis, dir);
								glm::mat4 rotation_matrix = glm::rotate(glm::mat4(1.0f), angle, dir);

								glm::vec4 verta4 = glm::vec4(scale * vs[t->a].position + offset, 1.0f) * rotation_matrix;
								glm::vec3 verta = glm::vec3(verta4.x, verta4.y, verta4.z);
								glm::vec4 vertb4 = glm::vec4(scale * vs[t->b].position + offset, 1.0f) * rotation_matrix;
								glm::vec3 vertb = glm::vec3(vertb4.x, vertb4.y, vertb4.z);
								glm::vec4 vertc4 = glm::vec4(scale * vs[t->c].position + offset, 1.0f) * rotation_matrix;
								glm::vec3 vertc = glm::vec3(vertc4.x, vertc4.y, vertc4.z);



								bool intersection_detection = glm::intersectRayTriangle(ray->position, ray->direction, verta, vertb, vertc, uv, d);
								if (intersection_detection) {
									glm::vec3 intersect = (d * direction) + position;
									float tr = d;//(intersect - position).length();
									if (tr < last_leng && tr >= 0.01f) {
										//printf("Intersection true for model %i!\n", g->model_index);

										intersect = ray->position + tr * ray->direction;
										glm::vec3 diff = intersect - position;

										//bayo_coord = glm::clamp(bayo_coord, 0.0f, 1.0f);

										glm::mat3 a = glm::mat3(glm::vec3(vs[t->a].uv, 1.0f), glm::vec3(vs[t->b].uv, 1.0f), glm::vec3(vs[t->c].uv, 1.0f));

										glm::mat3 a_inv = glm::inverse(a);
										glm::vec3 barycentric = calculateBarycentric(uv, vs[t->a].uv, vs[t->b].uv, vs[t->c].uv);

										//printf("Barycentric Coords: {%.2f, %.2f, %.2f}\n", barycentric.x, barycentric.y, barycentric.z);

										//uv = (barycentric.x * vs[t->a].uv + barycentric.y * vs[t->b].uv + barycentric.z * vs[t->c].uv) * uv;
										uv = uv;
										//uv = glm::vec2(barycentric.x, barycentric.y);

										//uv /= 10.0f;

										uv = glm::clamp(uv, 0.0f, 1.0f);

										//printf("UV Coords: {%.2f, %.2f}\n", uv.x, uv.y);
										ray->payload.color = glm::vec4(1.0f);
										ray->payload.intersection = intersect;
										ray->payload.uv = uv;
										ray->intersected = true;
										ray->payload.model = model;
										ray->payload.triangle = t;
										intersected = true;
										last_leng = tr;
									}
									//break;
								}
							}
						}
					}
				contin:;
					node = &model->bvh.nodes[next];
					continue;
				}
			}
			else {
				continue;
			}
			if (!intersected) {
				ray->payload.color = glm::vec4(0.0f, 0.0f, 0.0f, 1.0f);
				ray->intersected = false;
			}
		}
	}
}

__global__
void calculate_lighting(d_AmbientLight* amb, d_PointLight* lights, d_Material* materials, uint32_t lights_size, Ray* rays, glm::ivec2 dims, glm::vec4* out) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	uint32_t idx = y * dims.x + x;

	out[idx] = glm::vec4(0.0f);

	Ray* ray = &rays[idx];
	if (ray->intersected) {
		glm::vec3 result = glm::vec3(0.0f);
		glm::vec4 diffuse_color = sample_texture(materials[ray->payload.model->material_index].color_map.data, materials[ray->payload.model->material_index].color_map.dims, ray->payload.uv.x, ray->payload.uv.y);

		glm::vec4 lighting = glm::vec4(0.0f);
		bool lit = false;
		float cummulative_intensity = 0.0f;
		//printf("Light Count = %i\n", static_cast<int>(lights_size));
		for (int l = 0; l < static_cast<int>(lights_size); l++) {

			float amb_strength = 1.0f;
			glm::vec3 ambient = amb->intensity * amb->diffuse_color * amb_strength;
			glm::vec3 light_direction = lights[l].position - ray->payload.intersection;
			float distance = glm::length(light_direction);
			if (distance <= lights[l].range) {
				float diff = glm::max(glm::dot(ray->payload.triangle->normal, light_direction), 0.0f);
				float intensity = lights[l].intensity / (distance * distance);
				glm::vec3 diffuse = intensity * lights[l].diffuse_color * diff;

				result += (ambient + diffuse) + diffuse_color.w * glm::vec3(diffuse_color.x, diffuse_color.y, diffuse_color.z);

				cummulative_intensity += intensity;
			}

		}

		out[idx] = glm::vec4(result, 1.0f);
		out[idx] = glm::clamp(out[idx], 0.0f, 1.0f);
	}
	else {
		out[idx] = glm::vec4(0.32f, 0.32f, 0.1f, 1.0f);
	}
}

__global__
void texture_map(Ray* rays, glm::ivec2 dims, glm::vec4* out) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	uint32_t idx = y * dims.x + x;

	Ray* ray = &rays[idx];
	if (ray->intersected) {
		//out[idx] = sample_texture(ray->payload.model->color_map->data, ray->payload.model->color_map->dims, ray->payload.uv.x, ray->payload.uv.y);
	}
	else {
		out[idx] = ray->payload.color;
	}
}

__global__
void draw_crosshair(glm::ivec2 dims, glm::vec4* buffer, Crosshair cross, bool is_run, bool is_walk) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	int32_t idx = y * dims.x + x;

	x -= static_cast<int32_t>(static_cast<float>(dims.x) * 0.5f);
	y -= static_cast<int32_t>(static_cast<float>(dims.y) * 0.5f);

	float f_x = static_cast<float>(x) / static_cast<float>(dims.x), f_y = static_cast<float>(y) / static_cast<float>(dims.y);

	if (is_walk) {
		if (sqrtf(f_x * f_x + f_y * f_y) <= cross.walk_radius && sqrtf(f_x * f_x + f_y * f_y) > cross.walk_radius * 0.5f) {
			buffer[idx] = glm::vec4(1.0f);
		}
	}
	else if (is_run) {
		if (sqrtf(f_x * f_x + f_y * f_y) <= cross.run_radius && sqrtf(f_x * f_x + f_y * f_y) > cross.walk_radius * 0.5f) {
			buffer[idx] = glm::vec4(1.0f);
		}
	}
	else {
		buffer[idx] = buffer[idx];
	}
}

__global__
void fxaa_pass(glm::ivec2 dims, glm::vec4* buffer, glm::vec4* buffer_out) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	int32_t idx = y * dims.x + x;

	if (x > 0 && x < dims.x - 1 && y > 0 && y < dims.y - 1) {
		glm::vec4 pixels[8], current = glm::clamp(buffer[idx], 0.001f, 1.0f);
		float* out_matrix = new float[8];
		pixels[0] = buffer[idx - (dims.x) - 1];
		pixels[1] = buffer[idx - dims.x];
		pixels[2] = buffer[idx - dims.x + 1];

		pixels[3] = buffer[idx - 1];
		pixels[4] = buffer[idx + 1];

		pixels[5] = buffer[idx + dims.x - 1];
		pixels[6] = buffer[idx + dims.x];
		pixels[7] = buffer[idx + dims.x + 1];
		
		float total = 0.0f;
		for (uint8_t k = 0; k < 8; k++) {
			pixels[k] = glm::clamp(pixels[k], 0.001f, 1.0f);

			pixels[k] = pixels[k] / current;

			const float comp = 1.089f;

			if (pixels[k].x >= comp || pixels[k].y >= comp || pixels[k].z >= comp) {
				out_matrix[k] = glm::clamp(glm::dot(pixels[k], current), 0.0f, 1.0f);
				total += out_matrix[k];
			}
			else {
				out_matrix[k] = 1.0f;
			}
		}

		total /= 8.0f;

		//glm::mat3 matrix = glm::mat3(glm::vec3(out_matrix[0], out_matrix[3], out_matrix[5]), glm::vec3(out_matrix[1], 1.0f, out_matrix[7]), glm::vec3(out_matrix[2], out_matrix[4], out_matrix[7]));
		buffer_out[idx] = glm::clamp(total * buffer[idx], 0.0f, 1.0f);
	}
}

__global__
void copy_frame_buffer(glm::ivec2 dims, glm::vec4* dest, glm::vec4* from) {
	int j = blockDim.y * blockIdx.y + threadIdx.y,
		i = blockDim.x * blockIdx.x + threadIdx.x,
		x = (j * 128 + i) % dims.x,
		y = ((j * 128 + i) - x) / dims.x;
	int32_t idx = y * dims.x + x;

	if (x >= 0 && x < dims.x && y >= 0 && y < dims.y) {
		dest[idx] = from[idx];
	}
}